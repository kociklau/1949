#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <ctime>
#include <math.h>       /* pow, ceil */
#include <algorithm>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
//Windows has <direct.h>, POSIX systems have <unistd.h>
#include <unistd.h> /*To get the path to this script's directory*/
#include <sys/syslimits.h>

using namespace std;

__global__ void bench_Overhead(unsigned int *A, unsigned int d_tvalue[]){
    __shared__ unsigned int s_tvalue[1];
    s_tvalue[0] = A[0]; //Cold cache miss
    clock_t start_time = clock();
    s_tvalue[0] = A[0]; //Cache hit
    s_tvalue[0] = s_tvalue[0];//Dependency
    clock_t end_time = clock();
    s_tvalue[0] = end_time - start_time;
    d_tvalue[0] = s_tvalue[0];
}

int main()
{
	unsigned int *A = new unsigned int[3]; 
    unsigned int *h_tvalue = new unsigned int[1];
    const int iterations = 1000;
    unsigned int *avg_tval = new unsigned int[iterations];
	//Initialize array
    A[0] = 0x00;
    A[1] = 0x01;
    A[2] = 0x02;
   
    unsigned int *CUDA_A = new unsigned int[3];
    unsigned int *d_tvalue = new unsigned int[1];

    hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	//Places array into cache
	cudaStatus = hipMalloc((void**)&CUDA_A, 3 * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipDeviceReset(); //Clear all allocations and exit
	}

	//Places array into cache
	cudaStatus = hipMalloc((void**)&d_tvalue, sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for the tvalues array!");
		hipDeviceReset(); //Clear all allocations and exit
		return -1;
	}

    cudaStatus = hipMemcpy(CUDA_A, A, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for the array!");
		hipDeviceReset();
		return -1;
	}

    for (int i=0; i < iterations; i++){

        bench_Overhead<<<1,1>>>(CUDA_A, d_tvalue);

	    // Check for any errors launching the kernel
	    cudaStatus = hipGetLastError();
	    if (cudaStatus != hipSuccess) {
	    	fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	    	return -1;
	    }

	    // cudadevicesynchronize waits for the kernel to finish, and returns
	    // any errors encountered during the launch.
	    cudaStatus = hipDeviceSynchronize();
	    if (cudaStatus != hipSuccess) {
	    	fprintf(stderr, "cudadevicesynchronize returned error code %d after launching kernel!\n", cudaStatus);
	    	return -1;
	    }

        // Copy output vector from GPU buffer to host memory.
	    cudaStatus = hipMemcpy(h_tvalue, d_tvalue, sizeof(unsigned int), hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) {
	    	fprintf(stderr, "hipMemcpy failed! Could not retrieve tvalue from device.\n");
	    	return -1;
	    }

        //printf("overhead = %d\n",h_tvalue[0]);
        avg_tval[i] = h_tvalue[0];
    }

    //Print average of all integer accesses
    int sum = 0;
    for (int i=0; i<iterations; i++){
        sum = sum + avg_tval[i];
    }
    int avg = sum/iterations;
    printf("avg = %d\n",avg);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

