#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <ctime>
#include <math.h>       /* pow, ceil */
#include <algorithm>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
//Windows has <direct.h>, POSIX systems have <unistd.h>
#include <unistd.h> /*To get the path to this script's directory*/
#include <sys/syslimits.h>

using namespace std;

const int C = 6144;
//const int b = 8;//To cause misses.
const int b = 0;//For no misses.
//const int N = C+b+1; //24580 bytes = 24KB + 4B in binary => first miss
const int N = 9*8 + C+(b*5);
//const int N = 6144; //24576 bytes = 24KB => no miss => C = 6144
//const int N = 6000; //24000 bytes = 24KB in decimal => no miss
const int s = 8;//s = 128*4 bytes

//Max iterations with stored array 
//indeces being ints 
//const int iterations = 0xc000/8 - 4;

//34 because 17*500 = 8500 => traverse array N once. 
//*2 for traversing twice to eliminate cold miss effect.
//The modulus so it will do N+1 when odd and N+0 when even.
//Just so the division always ends up being an even # div by 2.
const int iterations = ((N+(N%2))/s)*4;
//const int iterations = ((N+(N%2))/s);
//const int iterations = 6143;
//const int iterations = 768*2;
//Both threshold and tolerance
//dynamically allocated in main()
unsigned int threshold = 200; float tolerance = 0.3; //30% tolerance
const char *path = getcwd(NULL,0);

__global__ void bench_CacheAccess(unsigned int *CUDA_A, unsigned int device_tvalue[], unsigned int device_index[]) {
	//Placing variables in shared memory makes them
	//not interfere with the global memory cache and, hence, the experiment
	__shared__ unsigned int s_tvalue[iterations];
	__shared__ unsigned int s_index[iterations];
	//__shared__ unsigned int s_tvalue[iterations];
	//__shared__ unsigned int s_index[iterations];
	//__shared__ int j;
    int j;
	j = 0;
	for (int it = 0; it < iterations; it++) {
		clock_t start_time = clock();
		j = CUDA_A[j];
		//Store the element index
		//Also generates memory dependence on previous
		//instruction, so that clock() happens after the
		//array access above
		s_index[it] = j;
		clock_t end_time = clock();
		//store the access latency
		s_tvalue[it] = end_time - start_time;
	}
	//All threads in this block have to reach this point
	//before continuing execution.
	__syncthreads();

	//Transfer results from shared memory to global memory
	//Later we will memcpy() the device global memory to host
	for (int i = 0; i < iterations; i++) {
		device_index[i] = s_index[i];
		device_tvalue[i] = s_tvalue[i];
	}

}

int main()
{
	printf("Will go through [%d] iterations with array of size N = [%d].\n", iterations, N);
	FILE * file;
	unsigned int *A = new unsigned int[N]; //The array of size N to test the cache 
	unsigned int *host_tvalue = new unsigned int[iterations]; //Time values for memory accesses
	unsigned int *host_index = new unsigned int[iterations]; //Index array of the accesses to the array elements
	int hits = 0, misses = 0;
	//Initialize array
	for (int i = 0; i < N; i++) {
		A[i] = (i + s) % N;
	}
	//Initialize index and time value arrays
	for (int k = 0; k < iterations; k++) {
		host_tvalue[k] = 0;
		host_index[k] = 0;
	}

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(bench_CacheAccess), hipFuncCachePreferL1);


	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	unsigned int *CUDA_A = 0; //When we allocate space for A on the GPU we assign it to this ptr, CUDA_A
	unsigned int *device_tvalue = 0; //Device variables needed to copy back to host.
	unsigned int *device_index = 0;


	//Places array into cache
	cudaStatus = hipMalloc((void**)&CUDA_A, N * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipDeviceReset(); //Clear all allocations and exit
	}

	//Places array into cache
	cudaStatus = hipMalloc((void**)&device_tvalue, iterations * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for the tvalues array!");
		hipDeviceReset(); //Clear all allocations and exit
		return -1;
	}

	//Places array into cache
	cudaStatus = hipMalloc((void**)&device_index, iterations * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for the index array!");
		hipDeviceReset(); //Clear all allocations and exit
		return -1;
	}

	for (int a = 0; a < 10; a++){
		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(CUDA_A, A, N * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for the array!");
			hipDeviceReset();
			return -1;
		}
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(device_index, host_index, iterations * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for the index array!");
		hipDeviceReset();
		return -1;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(device_tvalue, host_tvalue, iterations * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for the tvalues array!");
		hipDeviceReset();
	}

    // Classic P-chase benchmark.
	bench_CacheAccess<<<1,1>>>(CUDA_A, device_tvalue, device_index);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return -1;
	}

	// cudadevicesynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudadevicesynchronize returned error code %d after launching kernel!\n", cudaStatus);
		return -1;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(host_tvalue, device_tvalue, iterations * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! Could not retrieve tvalue from device.\n");
		return -1;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(host_index, device_index, iterations * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! Could not retrieve index from device.\n");
		return -1;
	}

	//Dynamically allocate threshold. First access is always a cold miss
	int delta = (int) (tolerance * host_tvalue[0]);
    //If I don't hardcode threshold, calculate it
	if (threshold == 0) threshold = host_tvalue[0] - delta;
	file = fopen("experiment_results.dat","w");
	if ((host_tvalue != NULL) && (host_index != NULL)) {
		for (int a = 0; a < iterations; a++) {
			if (host_tvalue[a] > threshold) misses++;
			else hits++;
		}
	}
	printf("%d hits\n%d misses\n", hits, misses);
	printf("threshold = %d\n", threshold);
    //printf("b = %d\n",((s*N)/(N-s*hits)));
	fprintf(file, "hits|misses\n");
	fprintf(file, "%d|%d\n", hits, misses);
	fprintf(file, "threshold=%d\n", threshold);
	fprintf(file, "arraySize=%d\n", N);
	fprintf(file, "stride=%d\n", s);
	fprintf(file, "numIterations=%d\n", iterations);
	fprintf(file,"arrayIndex|tvalue\n");
	for (int b = 0; b < iterations; b++) {
		//printf("host_index[%d] = %d\n",b,host_index[b]);
		//printf("host_tvalue[%d] = %d\n", b, host_tvalue[b]);
		fprintf(file,"%d|%d\n",host_index[b],host_tvalue[b]);
	}
	fprintf(file,"end\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	fclose(file);
	
	//Call python GUI script to show results
	printf("path = %s\n",path);
	//+7 because of "python\s"
	char cmd[PATH_MAX + 7];
	snprintf(cmd,sizeof(cmd),"python %s/../Python_Scripts/GUI.py", path);
	system(cmd);

    return 0;
}
